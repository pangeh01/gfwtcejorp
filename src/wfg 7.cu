#include "hip/hip_runtime.h"

/*
 * Author: Henry Peng (20165483)
 * University of Western Australia
 * School of Computer Science and Software Engineering
 * December 2010
 */

//#####################################
// Strategy for possible improvement:
//#####################################
// 1) Reduce Memcpy calls needed: Memcpy whole fr[iteration] to global variable ?
// 2) Reduce the number variables necessary for device (extra cudamallocing may reduce speed). e.g. remove temp variable
// 3) Parallel prefix sum, reduce number of threads needed? Can it increase speed? Currently: 256 threads
// 4) Reduce memory required for eliminated array and scan results array. Currently fixed at 512 points for each. plus Not scalable.
// 5) Reduce the size of memory allocated for each cuda variable.

// 6) Make into single kernel function with many device codes, single cudaThreadsynchronise (implement scan_best in device mode)
// 7) Look into using shared memory
//######################################

/////////////////////////////////////////////////////////
// Includes and Defines
/////////////////////////////////////////////////////////

//#define CUDPP_STATIC_LIB
#include "/usr/local/NVIDIA_GPU_Computing_SDK/C/common/inc/cudpp/cudpp.h"
#include "read.c"
#include "scan_best_kernel.cu"
#include <stdbool.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>
#include "cutil.h"
//#include "radixsort.cu"

//extern float ehv(int index, FRONT front);
//extern float hypervolume(FRONT front);

#define MIN(x, y) ((x < y) ? (x) : (y))
unsigned int frontSize;
unsigned int pointSize;

/////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////

//int maxDepth = -1;	//the maximum depth you have reached
int n = 0; //the dimension of the current front we are working on
int iteration = 0;	//depth of the recursion starting from 0
float hypervolume(FRONT);
FRONT *fr;	//storage for storing array of sprimed/non-dominating fronts as we go deeper into the recursion

/* Device global variables */
float *d_temp;
float *d_front;
int *d_eliminated;
int *d_scanoutput;
float *d_temp2;

// unused anymore
FRONT *frontsArray;
float *ehvStack;
float *hvStack;

// cpu memory stacks
int *indexStack;
int *nPointsStack;

// cuda memory stacks
float *d_frontsArray;
float *d_hvStack;
float *d_ehvStack;

// for sorting
int *prevOrder;
float *keys;
int *neworder;
int *used;

// allocate array for sorted results
float *d_out;

CUDPPHandle sortPlan;
CUDPPHandle scanPlan;

//NOte: n is needed for slicing and sorting, iteration is needed for saving array of fronts when going deeper into recursion

/////////////////////////////////////////////////////////
// GPU Kernel Functions
/////////////////////////////////////////////////////////

/**
 * Sprimes a front in parallel.
 */
__global__ void sprimeFront(float *frPoints_device, float *frontPoints_device, int index, int pointSize) {
	frPoints_device[blockIdx.x*pointSize+threadIdx.x] = MIN(frontPoints_device[index*pointSize+threadIdx.x], 
	frontPoints_device[(blockIdx.x+1+index)*pointSize+threadIdx.x]);
}

/**
 * Device Function: Determine domination status of point A and B.
 * Similar to CPU implementation.
 *
 * returns 1 if point b dominates a
 * zero if non-dominating
 * returns -1 if point a dominates b
 * returns 2 if point is equal
 */
__device__ int dominated(float *point_a, float *point_b, int nDim) {
	int result = 2;
	for (int i = 0; i < nDim; i++) {
		if (point_a[i] > point_b[i]) {
			if (result != 1) result = -1; else return 0;
		} 
		if (point_b[i] > point_a[i]) {
			if (result != -1) result = 1; else return 0;
		}
	}
	return result;
}

/**
 * Computes eliminated array in parallel.
 * Flag = 0 indicates eliminated, flag = 1 is kept.
 * e.g. result of a front with 5 points: [0, 1, 1, 0, 1]. 
 * (known "trivial" bug: equal points will not be eliminated). 
 */
__global__ void computeEliminatedArray(float *d_fr_iteration, int nDim, int *eliminated, int pointSize) {
    	__shared__ int flag;

    	flag = 1;
	__syncthreads();

	if (dominated(&d_fr_iteration[blockIdx.x*pointSize] , &d_fr_iteration[threadIdx.x*pointSize], nDim) == 1)
		flag = 0;

	__syncthreads();
    
    	eliminated[blockIdx.x] = flag;
}

/**
 * Insert the results and reorder into temp array in parallel.
 */ 
__global__ void insertResults(float *d_fr_iteration, float *temp, int *eliminated, int *scanoutput, int pointSize) {
	if (eliminated[blockIdx.x] == 1) {
		//insert the non-dominated points
		temp[(scanoutput[blockIdx.x]-1)*pointSize+threadIdx.x] = d_fr_iteration[blockIdx.x*pointSize+threadIdx.x];
	}
}


////////////////////////////////////////////////////////////////
// Start of CUDA CODE
////////////////////////////////////////////////////////////////

/**
 * Returns a sprimed & non-dominating front relative to point p at index.
 */
void limitset() {
	// sets the number of points in sprimed front
	int z = nPointsStack[iteration-1] - 1 - indexStack[iteration-1];

	// sprimes the front and store it into temporary storage
	sprimeFront<<< z, n >>>( d_temp, &d_frontsArray[frontSize*(iteration-1)], indexStack[iteration-1], pointSize);
    	//hipDeviceSynchronize();

	// compute eliminated array and store it in d_eliminated
	computeEliminatedArray<<< z, z >>>(d_temp, n, d_eliminated, pointSize);
	//hipDeviceSynchronize();
	
	// Run the scan
	cudppScan(scanPlan, d_scanoutput, d_eliminated, z);
	//hipDeviceSynchronize();

	// compute the results and store it in frontArray
	insertResults<<<z,n>>> (d_temp, &d_frontsArray[frontSize*iteration], d_eliminated, d_scanoutput, pointSize);
	//hipDeviceSynchronize();

	// update number of points to the host
	hipMemcpy(&nPointsStack[iteration], &d_scanoutput[z-1], sizeof(int), hipMemcpyDeviceToHost); //update number of points
}

void setUpPlan() {
	// set the config
	CUDPPConfiguration config;
	//config.op = CUDPP_MAX;
	config.datatype = CUDPP_FLOAT;
	config.algorithm = CUDPP_SORT_RADIX_GLOBAL;
	config.options = CUDPP_OPTION_FORWARD;
	
	// create the plan
	cudppPlan(&sortPlan, config, frontSize/pointSize, 1, 0);

    	//config.op = CUDPP_ADD;
    	config.datatype = CUDPP_INT;
    	config.algorithm = CUDPP_SCAN;
   	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
    
   	CUDPPResult result = cudppPlan(&scanPlan, config, frontSize/pointSize, 1, 0); 
}

__global__ void initialise(int *prevOrder) {
	prevOrder[threadIdx.x] = threadIdx.x;
}

__global__ void initialiseKeys(float *keys, float *d_in, int i, int pointSize) {
	keys[threadIdx.x] = d_in[threadIdx.x*pointSize+i];
}

__global__ void initialiseUsed(int *used) {
	used[threadIdx.x] = -1;
}

__global__ void setuporder(int *used, float *d_in, int *prevOrder, float *keys, int numElements, int i, int pointSize, int *neworder) {
	for (int j = 0; j < numElements; j++)    {
		for (int k = 0; k < numElements; k++)    {
	   		
			if (used[k] == -1 && d_in[prevOrder[k]*pointSize+i] == keys[j]){
				neworder[j] = prevOrder[k];
				used[k] = 0;
				break;
	    		}
		}
	}

}

__global__ void arrange(float *d_out, float *d_in, int *prevOrder) {

	d_out[threadIdx.x*gridDim.x+blockIdx.x] = d_in[prevOrder[threadIdx.x]*gridDim.x+blockIdx.x];
	
}

void sortPoints(float *d_in, int numElements) {

	initialise<<<1,numElements>>>(prevOrder);

	for (int i = 0; i < n; i++) {
		initialiseKeys<<<1, numElements>>>(keys, d_in, i, pointSize);
	   	initialiseUsed<<<1, numElements>>>(used);

		cudppSort(sortPlan, keys, keys, numElements);

		setuporder<<<1,1>>>(used, d_in, prevOrder, keys, numElements, i, pointSize, neworder);

		hipMemcpy(prevOrder, neworder, sizeof(int)*numElements, hipMemcpyDeviceToDevice);
        }

	//arrange the order according to the last objectives
	arrange<<< pointSize, numElements>>>(d_out, d_in, prevOrder);

	// copy d_out back to d_in
	hipMemcpy(d_in, d_out, numElements*pointSize*sizeof(float), hipMemcpyDeviceToDevice);
}


//////////////////////////////////////////////////////////
//  HV CUDA
//////////////////////////////////////////////////////////

__global__ void parallel_multiply(float *d_ehvStack, float *d_frontsArray, int iteration, int frontSize, int pointSize, int index, int n) {
	d_ehvStack[iteration] = 1;

	for (int i = 0; i < n; i++)  {
		d_ehvStack[iteration] *= d_frontsArray[frontSize*iteration+pointSize*index+i];
	}
}

__global__ void multiply2(float *d_hvStack, float *d_frontsArray, int iteration, int frontSize, int pointSize, int index, float *d_ehvStack, int n) {
	d_hvStack[iteration] = d_frontsArray[frontSize*iteration+pointSize*index+n] * d_ehvStack[iteration];
}

__global__ void compute2d(float *d_ehvStack, int iteration, float *d_frontsArray, int pointSize, int nPoints, int frontSize, float *d_hvStack, int index, int n ) {
	d_ehvStack[iteration] -= d_frontsArray[frontSize*(iteration+1)+pointSize*0+0] * d_frontsArray[frontSize*(iteration+1)+pointSize*0+1]; 
	for (int i = 1; i < nPoints; i++) {
		d_ehvStack[iteration] -= d_frontsArray[frontSize*(iteration+1)+pointSize*i+0] * 
					(d_frontsArray[frontSize*(iteration+1)+pointSize*i+1] - d_frontsArray[frontSize*(iteration+1)+pointSize*(i-1)+1]);
	}
	d_hvStack[iteration] += d_frontsArray[frontSize*iteration+pointSize*index+n] * d_ehvStack[iteration];
}

__global__ void computeFinishedLevel(float *d_ehvStack, float *d_hvStack, int iteration, float *d_frontsArray, int frontSize, int pointSize, int n, int index) {
	d_ehvStack[iteration] -= d_hvStack[iteration+1]; 
      	d_hvStack[iteration] += d_frontsArray[frontSize*iteration+pointSize*index+n] * d_ehvStack[iteration];
}

void hvparallel() {  
	setUpPlan();

	// sort the array
	sortPoints(&d_frontsArray[frontSize*0], nPointsStack[0]); // sorts the points located in front[0], use nPointsStack[0] for the number of points

	indexStack[0] = nPointsStack[0] - 1;

	while (indexStack[0] >= 0) {
		if (indexStack[iteration] < 0) {
			iteration--; 
			computeFinishedLevel<<<1,1>>>(d_ehvStack, d_hvStack, iteration, d_frontsArray, frontSize, pointSize, n, indexStack[iteration]);
       			indexStack[iteration]--;
       			n++;
		} else if (n == 2) {
			iteration--;
       			compute2d<<<1, 1 >>>(d_ehvStack, iteration, d_frontsArray, pointSize, nPointsStack[iteration+1], frontSize, d_hvStack, indexStack[iteration], n);
      			indexStack[iteration]--;
       			n++;
		} else {
      			n--;

       			parallel_multiply<<< 1, 1>>>(d_ehvStack, d_frontsArray, iteration, frontSize, pointSize, indexStack[iteration], n);

       			if (indexStack[iteration] == nPointsStack[iteration] - 1) {
				multiply2<<<1, 1>>>(d_hvStack, d_frontsArray, iteration, frontSize, pointSize, indexStack[iteration], d_ehvStack, n);
          			indexStack[iteration]--;
          			n++;
			} else {
         			iteration++;
          			limitset(); 
          			sortPoints(&d_frontsArray[frontSize*iteration], nPointsStack[iteration]);
          			indexStack[iteration] = nPointsStack[iteration]-1;
			}
		}
	}

}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////

int main(int argc, char *argv[]) {
	//CUT_DEVICE_INIT(argc, argv);
	
	// read the file
	FILECONTENTS *f = readFile(argv[1]);
	
	// start the timer
	struct timeval tv1, tv2;
	struct rusage ru_before, ru_after;
	getrusage (RUSAGE_SELF, &ru_before);
	
	int maxDimensions = 0;	//the max number of dimensions in the fronts
	int maxPoints = 0;  //the max number of points in the fronts
	
	// find the max number of Points, and the max number of Dimensions
	for (int i = 0; i < f->nFronts; i++) {
		if (f->fronts[i].nPoints > maxPoints) 
			maxPoints = f->fronts[i].nPoints;
		if (f->fronts[i].n > maxDimensions) 
			maxDimensions = f->fronts[i].n;
  	}

	/* allocate for cuda memory */
	hipMalloc( (void **) &d_temp, maxPoints*maxDimensions*sizeof(float));
	hipMalloc((void **) &d_scanoutput, (512)*sizeof(int));
	hipMalloc((void **) &d_eliminated, (512)*sizeof(int));
	hipMalloc( (void **) &d_front, maxPoints*maxDimensions*sizeof(float));
	hipMalloc((void**) &d_temp2, sizeof(float)*maxPoints*maxDimensions);
	
	// allocate cuda memory
	frontSize = maxPoints*maxDimensions;
	pointSize = maxDimensions;
	hipMalloc((void **) &d_frontsArray, frontSize * maxDimensions * sizeof(float));
	hipMalloc((void **) &d_ehvStack, sizeof(float) * maxDimensions);
	hipMalloc((void **) &d_hvStack, sizeof(float) * maxDimensions);

	// allocate cuda memory
	hipMalloc( (void **) &prevOrder, maxPoints*sizeof(int));
	hipMalloc( (void **) &keys, maxPoints*sizeof(float));
	hipMalloc( (void **) &neworder, maxPoints*sizeof(int));
	hipMalloc( (void **) &used, maxPoints*sizeof(int));

	hipMalloc( (void **) &d_out, maxPoints*sizeof(float));

	// allocate cpu memory Stacks
	indexStack = (int *) malloc(sizeof(int) * maxDimensions);
	nPointsStack = (int *) malloc(sizeof(int) * maxDimensions);

	// process each front to get the hypervolumes
	for (int i = 0; i < f->nFronts; i++) {
		// read each front
		FRONT front = f->fronts[i];
		n = front.n;
		nPointsStack[0] = front.nPoints;

		// copy front to device memory
		float h_front[front.nPoints*pointSize]; 
		for (int j = 0; j < front.nPoints; j++) {
			for (int k = 0; k < n; k++) {
				h_front[j*pointSize+k] = front.points[j].objectives[k];
			}
		}
		hipMemcpy(d_frontsArray, h_front, frontSize*sizeof(float), hipMemcpyHostToDevice);		

		// run hv parallel
		hvparallel();

 		// copy back hvresult
		float *hvResult = (float *) malloc(sizeof(float));
		hipMemcpy(hvResult, d_hvStack, sizeof(float), hipMemcpyDeviceToHost);

		// print them out
		printf("Calculating Hypervolume for Front:%d...\n", i+1);
		printf("\t\t\t\t\t%f\n", hvResult[0]);
	}
	
	// stop timer
	getrusage (RUSAGE_SELF, &ru_after);
	tv1 = ru_before.ru_utime;
	tv2 = ru_after.ru_utime;
	printf("Average time = %fs\n", (tv2.tv_sec + tv2.tv_usec * 1e-6 - tv1.tv_sec - tv1.tv_usec * 1e-6) / f->nFronts);

	// TODO free the storage

	return 0;
}

//////////////////////////////////
// HELPER METHODS
//////////////////////////////////

/**
 *  Timer Functions
 */
void run(int argc, char *argv[])
{
    unsigned int timer = 0;

    CUT_DEVICE_INIT(argc, argv);

    /////////////////////////////////////////////////////////////////////
    // Create and start a timer called "timer"
    // alls to create ans start times are enveloped in the CUT_SAFE_CALL
    // This CUDA Utility Tool checks for errors upon return.
    // If an error is found, it prints out and error message, file name,
    // and line number in file where the error can be found
    /////////////////////////////////////////////////////////////////////
    timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));
    
    // Stop the timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue(timer));

    // Delete the timer
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}

/**
 * Prints a front.
 */
void printfront(FRONT front) {
	for (int j = 0; j < front.nPoints; j++)
	{
		printf("\t");
		for (int k = 0; k < n; k++)
		{
			printf("%f ",front.points[j].objectives[k]);
		}
		printf("\n");
	}
}

/**
 * prints a front located on device
 */
void printfront(float *d_front, int numPoints) {
	printf("----------------------------------\n");
	float *front = (float *) malloc(frontSize*sizeof(float));
	hipMemcpy(front, d_front, frontSize*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < numPoints; i++) {
		for (int j = 0; j < n; j++) {
			printf("%1.1f ", front[i*pointSize+j]);
		}
		printf("\n");
	}
	printf("----------------------------------\n");
	free(front);
}

void printElements(float *elements, int numElements) {
	float *sup = (float *) malloc(sizeof(float)*numElements);
	hipMemcpy(sup, elements, sizeof(float)*numElements, hipMemcpyDeviceToHost);
	for (int i = 0; i < numElements; i++) {
		printf("%f ", sup[i]);
	}
	printf("\n");
}

void printElements(int *elements, int numElements) {
	int *sup = (int *) malloc(sizeof(int)*numElements);
	hipMemcpy(sup, elements, sizeof(int)*numElements, hipMemcpyDeviceToHost);
	for (int i = 0; i < numElements; i++) {
		printf("%d ", sup[i]);
	}
	printf("\n");
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
        exit(-1);
    }                         
}

