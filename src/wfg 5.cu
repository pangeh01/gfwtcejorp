#include "hip/hip_runtime.h"

/*
 * Author: Henry Peng (20165483)
 * University of Western Australia
 * School of Computer Science and Software Engineering
 * December 2010
 */

//#####################################
// Strategy for possible improvement:
//#####################################
// 1) Reduce Memcpy calls needed: Memcpy whole fr[iteration] to global variable ?
// 2) Reduce the number variables necessary for device (extra cudamallocing may reduce speed). e.g. remove temp variable
// 3) Parallel prefix sum, reduce number of threads needed? Can it increase speed? Currently: 256 threads
// 4) Reduce memory required for eliminated array and scan results array. Currently fixed at 512 points for each. plus Not scalable.
// 5) Reduce the size of memory allocated for each cuda variable.

// 6) Make into single kernel function with many device codes, single cudaThreadsynchronise (implement scan_best in device mode)
// 7) Look into using shared memory
//######################################

/////////////////////////////////////////////////////////
// Includes and Defines
/////////////////////////////////////////////////////////

//#define CUDPP_STATIC_LIB
#include "/usr/local/NVIDIA_GPU_Computing_SDK/C/common/inc/cudpp/cudpp.h"
#include "read.c"
#include "scan_best_kernel.cu"
#include <stdbool.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>
#include "cutil.h"
//#include "radixsort.cu"

//extern float ehv(int index, FRONT front);
//extern float hypervolume(FRONT front);

#define MIN(x, y) ((x < y) ? (x) : (y))
unsigned int frontSize;
unsigned int pointSize;

/////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////

//int maxDepth = -1;	//the maximum depth you have reached
int n = 0; //the dimension of the current front we are working on
int iteration = 0;	//depth of the recursion starting from 0
float hypervolume(FRONT);
FRONT *fr;	//storage for storing array of sprimed/non-dominating fronts as we go deeper into the recursion

/* new hypervolume variables */
FRONT *fs;      // the stack of fronts
int *ps;        // the indices of the contributing points
float *vs;     // partial volumes
float *evs;    // exclusive volumes

/* Device global variables */
float *d_temp;
float *d_front;
int *d_eliminated;
int *d_scanoutput;
float *d_temp2;

/* new hypervolume variables */
FRONT *frontsArray;      // the stack of fronts
int *indexStack;        // the indices of the contributing points
float *hvStack;     // partial volumes
float *ehvStack;    // exclusive volumes

/* new cuda variables */
FRONT *d_frontsArray;
float *d_hvStack;
float *d_ehvStack;

//NOte: n is needed for slicing and sorting, iteration is needed for saving array of fronts when going deeper into recursion

/////////////////////////////////////////////////////////
// GPU Kernel Functions
/////////////////////////////////////////////////////////

/**
 * Sprimes a front in parallel.
 */
__global__ void sprimeFront(float *frPoints_device, float *frontPoints_device, int index) {
	frPoints_device[blockIdx.x*blockDim.x+threadIdx.x] = MIN(frontPoints_device[index*blockDim.x+threadIdx.x], 
	frontPoints_device[(blockIdx.x+1+index)*blockDim.x+threadIdx.x]);
}

/**
 * Device Function: Determine domination status of point A and B.
 * Similar to CPU implementation.
 *
 * returns 1 if point b dominates a
 * zero if non-dominating
 * returns -1 if point a dominates b
 * returns 2 if point is equal
 */
__device__ int dominated(float *point_a, float *point_b, int nDim) {
	int result = 2;
	for (int i = 0; i < nDim; i++) {
		if (point_a[i] > point_b[i]) {
			if (result != 1) result = -1; else return 0;
		} 
		if (point_b[i] > point_a[i]) {
			if (result != -1) result = 1; else return 0;
		}
	}
	return result;
}

/**
 * Computes eliminated array in parallel.
 * Flag = 0 indicates eliminated, flag = 1 is kept.
 * e.g. result of a front with 5 points: [0, 1, 1, 0, 1]. 
 * (known "trivial" bug: equal points will not be eliminated). 
 */
__global__ void computeEliminatedArray(float *d_fr_iteration, int nDim, int *eliminated) {
    	__shared__ int flag;

    	flag = 1;
	__syncthreads();

	if (dominated(&d_fr_iteration[blockIdx.x*nDim] , &d_fr_iteration[threadIdx.x*nDim], nDim) == 1)
		flag = 0;

	__syncthreads();
    
    	//if (threadIdx.x==0) {
    	    eliminated[blockIdx.x] = flag;
	//}
}

/**
 * Insert the results and reorder into temp array in parallel.
 */ 
__global__ void insertResults(float *d_fr_iteration, float *temp, int *eliminated, int *scanoutput) {
	if (eliminated[blockIdx.x] == 1) {
		//insert the non-dominated points
		temp[(scanoutput[blockIdx.x]-1)*blockDim.x+threadIdx.x] = d_fr_iteration[blockIdx.x*blockDim.x+threadIdx.x];
	} /*else {
		//if eliminated insert at the end of the temp array.
		temp[(gridDim.x-1-(blockIdx.x-scanoutput[blockIdx.x]))*blockDim.x+threadIdx.x] = d_fr_iteration[blockIdx.x*blockDim.x+threadIdx.x];
	}*/
}

/**
 * Create an inclusive scan output from exclusive scan output.
 * Shift array left, and insert the sum of last element of scan and 
 * last element of input array, at the end of the sum.
 */ 
__global__ void scan_inclusive(int *d_scanbest, int *d_eliminated, int nPoints) {
	if (threadIdx.x > 0) 
		d_scanbest[threadIdx.x-1] = d_scanbest[threadIdx.x];
	
	//__syncthreads();
	//if (threadIdx.x == nPoints-1) {
	if (nPoints == 1) {
		d_scanbest[nPoints-1] = d_eliminated[nPoints-1];
	} else {
		d_scanbest[nPoints-1] = d_scanbest[nPoints-2] + d_eliminated[nPoints-1];
	}
	//}
}

/////////////////////////////////////////////////////////
// Helper methods
/////////////////////////////////////////////////////////

/**
 * Prefix-sum sequential on CPU. (Deprecated)
 */
int *sequentialScan(int *eliminated, int n)
{
	int *output = (int *) malloc(sizeof(int) *n);
	output[0] = eliminated[0];
	for (int i = 1; i < n; i++) {
		output[i] = output[i-1] + eliminated[i];
	}
	return output;
}

/**
 * Prints a front.
 */
void printfront(FRONT front) {
	for (int j = 0; j < front.nPoints; j++)
	{
		printf("\t");
		for (int k = 0; k < n; k++)
		{
			printf("%f ",front.points[j].objectives[k]);
		}
		printf("\n");
	}
}

/////////////////////////////////////////////////////////
// CPU Functions
/////////////////////////////////////////////////////////

/**
 * Determine domination status of point A and B.
 *
 * returns 1 if point b dominates a
 * zero if non-dominating
 * returns -1 if point a dominates b
 * returns 2 if point is equal
 */
int dominated(POINT a, POINT b) {
	int result = 2;
	for (int i = 0; i < n; i++) {
		if (a.objectives[i] > b.objectives[i]) {
			if (result != 1) result = -1; else return 0;
		}
		if (b.objectives[i] > a.objectives[i]) {
			if (result != -1) result = 1; else return 0;
		}
	}
	return result;
}

/**
 * compare function for qsort sorting front in the last objective, i.e. increasing from top to bottom
 * and we process hypervolumes from the bottom 
 */
int compare (const void *a, const void *b)
{
	//n == maxDimensions-iteration
	for (int i = n - 1; i >= 0; i--) {
		if (((*(POINT *)a).objectives[i] > (*(POINT *)b).objectives[i])) return 1;
		if (((*(POINT *)a).objectives[i] < (*(POINT *)b).objectives[i])) return -1;
	}
	return 0;
}

/**
 * Returns the size of exclusive hypervolume of point p at index relative to a front set.
 */
float ehv(int index, FRONT front) {
	
	//hypervolume of a single poinit
	float ehv = 1;
	for (int i = 0; i < n; i++) {
		ehv *= front.points[index].objectives[i];
	}
	
	//if not the last point, then go deeper into the recursion
	if (index < front.nPoints-1) {
		limitset(index, front);		//limit the front relative to index.
		iteration++;	//slicing
		ehv -= hypervolume(fr[iteration-1]);	//subtract the hypervolume of the limit set from ehv.
		iteration--;
	}
	
	return ehv;
}

/**
 * Returns the size of hypervolume of a front.
 */
float hypervolume(FRONT front) {
	//sort the front with qsort
	qsort(front.points, front.nPoints, sizeof (POINT), compare);
	
	//calculate for base case = 2D
	if (n==2) {
		float vol2d = (front.points[0].objectives[0] * front.points[0].objectives[1]);
		for (int i = 1; i < front.nPoints; i++) {
			vol2d += (front.points[i].objectives[0]) * 
						   (front.points[i].objectives[1] - front.points[i - 1].objectives[1]);
		}
		return vol2d;
	}
	
	float sumhv = 0;
	n--;
	//sum all the segments
	for (int i = front.nPoints - 1; i >= 0; i--)
		//for (int i = 0; i < front.nPoints; i++) //annoying bug that cause inaccurate results
		sumhv += front.points[i].objectives[n] * ehv(i, front);
	n++;
	
	return sumhv;
}

// creates the front frontsArray[fr-1].points[indexStack[fr-1]+1 ..] in frontsArray[fr], 
// with each point bounded by frontsArray[fr-1].points[indexStack[fr-1]] 
// and with dominated points removed 
void makeDominatedBit()
{

  	int z = frontsArray[iteration-1].nPoints - 1 - indexStack[iteration-1];
	for (int i = 0; i < z; i++) {
		for (int j = 0; j < n; j++) {
			frontsArray[iteration].points[i].objectives[j] = MIN(frontsArray[iteration-1].points[indexStack[iteration-1]].objectives[j],
                                     frontsArray[iteration-1].points[indexStack[iteration-1] + 1 + i].objectives[j]);
		}
	}
  
	POINT t; // have to do proper swaps because of the reuse of the memory hierarchy 
  	frontsArray[iteration].nPoints = 1;
  	for (int i = 1; i < z; i++) {
		int j = 0;
   		bool keep = true;

    		while (j < frontsArray[iteration].nPoints && keep) {
       			switch (dominated(frontsArray[iteration].points[i], frontsArray[iteration].points[j])) {
				case -1: 
					t = frontsArray[iteration].points[j];
					frontsArray[iteration].points[j] = frontsArray[iteration].points[frontsArray[iteration].nPoints - 1]; 
					frontsArray[iteration].points[frontsArray[iteration].nPoints - 1] = t; 
					frontsArray[iteration].nPoints--; 
					break;
          			
				case  0: 
					j++; 
					break;
          
					// case  2: printf("Identical points!\n");
	 			default: 
					keep = false;
			}
		}
     		
		if (keep) {
			t = frontsArray[iteration].points[frontsArray[iteration].nPoints]; 
			frontsArray[iteration].points[frontsArray[iteration].nPoints] = frontsArray[iteration].points[i]; 
			frontsArray[iteration].points[i] = t; 
			frontsArray[iteration].nPoints++;	
		}
    	}
}

void hvnew() {
	// sets hvStack[0] to the hypervolume of frontsArray[0][0 ..] 
  	qsort(frontsArray[0].points, frontsArray[0].nPoints, sizeof(POINT), compare);

  	indexStack[0] = frontsArray[0].nPoints - 1;

  	while (indexStack[0] >= 0) { // there are jobs remaining 
    		if (indexStack[iteration] < 0) {	// we've finished the jobs at this level: i.e. completed all ehv calculation (HV is complete for that level!)
			iteration--; 
			// compute the single point ehv excluding the last objective
      			ehvStack[iteration] -= hvStack[iteration+1]; 
			//  add the ehv multiplied by the last objective left out due to n--, to the hv stack
      			hvStack[iteration] += (frontsArray[iteration].points[indexStack[iteration]].objectives[n]) * ehvStack[iteration];
			// 1 job is finished for the previous iteration
       			indexStack[iteration]--;
			// finished with next level ehv
       			n++;
		} else if (n == 2) {  	// do this job using the linear algorithm 
			//TODO make this work
			/*if (indexStack[0] == 0) { //or iteration== 0
				hvStack[0] = frontsArray[0].points[0].objectives[0] * frontsArray[0].points[0].objectives[1];
				for (int i = 1; i < frontsArray[0].nPoints; i++) {
					hvStack[0] += (frontsArray[0].points[i].objectives[0]) * 
								   (frontsArray[0].points[i].objectives[1] - frontsArray[0].points[i - 1].objectives[1]);
				}
				indexStack[0]--;
				n++;
			} else {*/
			iteration--;
       			ehvStack[iteration] -= frontsArray[iteration+1].points[0].objectives[0] * frontsArray[iteration+1].points[0].objectives[1]; 
       			for (int i = 1; i < frontsArray[iteration+1].nPoints; i++) {
         			ehvStack[iteration] -= (frontsArray[iteration+1].points[i].objectives[0]) * (frontsArray[iteration+1].points[i].objectives[1] - frontsArray[iteration+1].points[i-1].objectives[1]);
			}
       			hvStack[iteration] += frontsArray[iteration].points[indexStack[iteration]].objectives[n] * ehvStack[iteration];
      			indexStack[iteration]--;
       			n++;
			//}
		} else {  // we need to "recurse" 
      			n--;

       			ehvStack[iteration] = 1;
       			for (int i = 0; i < n; i++)  {
				//compute the single point ehv excluding the last objective
         			ehvStack[iteration] *= frontsArray[iteration].points[indexStack[iteration]].objectives[i];
			}

       			if (indexStack[iteration] == frontsArray[iteration].nPoints - 1) { 	// first job at this level: set will be empty = no need to recurse
				// add the first ehv multiplied by the last objective left out due to n--, to the hv stack
        			hvStack[iteration] = frontsArray[iteration].points[indexStack[iteration]].objectives[n] * ehvStack[iteration];
          			indexStack[iteration]--;
				// finished with first level ehv (index = nPoints-1), now need to calculate the levels until reach (index = 0)
          			n++;
			} else { // set will be non-empty: create a new job 
				//go to next level recursion
         			iteration++; 
          			makeDominatedBit(); 
          			qsort(frontsArray[iteration].points, frontsArray[iteration].nPoints, sizeof(POINT), compare);
				//reset index stack to the number of points-1
          			indexStack[iteration] = frontsArray[iteration].nPoints - 1;
			}
		}
	}
}

/**
 *  Timer Functions
 */
void run(int argc, char *argv[])
{
    unsigned int timer = 0;

    CUT_DEVICE_INIT(argc, argv);

    /////////////////////////////////////////////////////////////////////
    // Create and start a timer called "timer"
    // alls to create ans start times are enveloped in the CUT_SAFE_CALL
    // This CUDA Utility Tool checks for errors upon return.
    // If an error is found, it prints out and error message, file name,
    // and line number in file where the error can be found
    /////////////////////////////////////////////////////////////////////
    timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));
    
    // Stop the timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue(timer));

    // Delete the timer
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}

/**
 * Runs a parallel hypervolume
 */ 
__global__ void hvparallellol() {
	// Should call many device functions

	//sortParallel();

	//d_indexStack[0] = d_frontsArray[0].nPoints - 1;
	
}

/**
 * Runs a parallel hypervolume
 */
//void hvparallel() {
	/*int blockSize = 100;
	int nBlocks = N/blockSize + (N%blockSize == 0 ? 0:1);
	// where N is the parallel threads required

	global<<<nBlocks, blockSize>>> ( param , N );

	hipDeviceSynchronize();
	checkCUDAError("HV parallel failed!");*/
//}

////////////////////////////////////////////////////////////////
// CUDA Helpers
////////////////////////////////////////////////////////////////

/**
 * Returns a sprimed & non-dominating front relative to point p at index.
 */
void limitset() {
	// TODO make this a kernel which calls many device functions 

	// sets the number of points in sprimed front
	int z = frontsArray[iteration-1].nPoints - 1 - indexStack[iteration-1];
	
	// sprimes the front and store it into temporary storage
	sprimeFront<<< z, n >>>( d_temp, d_frontsArray, indexStack[iteration-1]);
    	hipDeviceSynchronize();

	// compute eliminated array and store it in d_eliminated
	computeEliminatedArray<<< z, z >>>(d_temp, n, d_eliminated);
	hipDeviceSynchronize();
	
	// compute parallel prefix sum and store the result in d_scanoutput
	// TODO may need to make use of cudpp for this
	scan_best<<< 256, 512/2, sizeof(int)*(512) >>>(d_scanoutput, d_eliminated, 512);
	hipDeviceSynchronize();
	scan_inclusive<<< 1, z >>>(d_scanoutput, d_eliminated, z);  //make the result into an inclusive scan result.
	hipDeviceSynchronize();

	// compute the results and store it in frontArray
	insertResults<<<z,n>>> (d_temp, &frontsArray[iteration], d_eliminated, d_scanoutput);
	hipDeviceSynchronize();

	// update number of points to the host
	hipMemcpy(&indexStack[iteration], &d_scanoutput[z-1], sizeof(int), hipMemcpyDeviceToHost); //update number of points
}

/**
 * @param front front to sort
 * @param numElements number of points
 * @param size size of each point
 */
void sortParallel(float *d_points, int numElements, int size) {
	// set the config
	CUDPPConfiguration config;
	config.op = CUDPP_ADD;
	config.datatype = CUDPP_FLOAT;
	config.algorithm = CUDPP_SORT_RADIX;
	config.options = CUDPP_OPTION_FORWARD;
	
	// create the plan
	CUDPPHandle sortPlan = 0;
	CUDPPResult result = cudppPlan(&sortPlan, config, numElements, 1, 0);  

	// if not successful then exit
	if (CUDPP_SUCCESS != result)
	{
		printf("Error creating CUDPPPlan\n");
		exit(-1);
	}

	// Run the sort TODO make the sorting works
	cudppSort(sortPlan, d_out, d_in, numElements);

	// Destroy the plan
	result = cudppDestroyPlan(scanplan);
	if (CUDPP_SUCCESS != result)
	{
		printf("Error destroying CUDPPPlan\n");
		exit(-1);
	}

	// TODO reuse config and destroy plan at the end
}

//////////////////////////////////////////////////////////
//  HV CUDA
//////////////////////////////////////////////////////////

void hvparallel(int nPoints) {
	// sort in parallel
	sortParallel(frontsArray[0].points, nPoints, sizeof(POINT));

	indexStack[0] = nPoints - 1;

	// TODO host cannot access device memory ehv, and hv and frontsArray, need CUDA kernels for this
	while (indexStack[0] >= 0) {
		if (indexStack[iteration] < 0) {
			iteration--; 
      			ehvStack[iteration] = ehvStack[iteration] - hvStack[iteration+1]; 
      			hvStack[iteration] = hvStack[iteration] + (frontsArray[iteration].points[indexStack[iteration]].objectives[n]) * ehvStack[iteration];
       			indexStack[iteration]--;
       			n++;
		} else if (n == 2) {
			iteration--;
       			ehvStack[iteration] -= frontsArray[iteration+1].points[0].objectives[0] * frontsArray[iteration+1].points[0].objectives[1]; 
       			for (int i = 1; i < frontsArray[iteration+1].nPoints; i++) {
         			ehvStack[iteration] -= (frontsArray[iteration+1].points[i].objectives[0]) * 
							(frontsArray[iteration+1].points[i].objectives[1] - frontsArray[iteration+1].points[i-1].objectives[1]);
			}
       			hvStack[iteration] += frontsArray[iteration].points[indexStack[iteration]].objectives[n] * ehvStack[iteration];
      			indexStack[iteration]--;
       			n++;
		} else {
      			n--;
       			ehvStack[iteration] = 1;
       			for (int i = 0; i < n; i++)  {
         			ehvStack[iteration] *= frontsArray[iteration].points[indexStack[iteration]].objectives[i];
			}

       			if (indexStack[iteration] == frontsArray[iteration].nPoints - 1) {
        			hvStack[iteration] = frontsArray[iteration].points[indexStack[iteration]].objectives[n] * ehvStack[iteration];
          			indexStack[iteration]--;
          			n++;
			} else {
         			iteration++; 
          			makeDominatedBit(); 
          			sortParallel(frontsArray[iteration].points, indexStack[iteration], sizeof(POINT));
          			indexStack[iteration]--;
			}
		}
	}
}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////

int main(int argc, char *argv[]) {
	//CUT_DEVICE_INIT(argc, argv);
	
	// read the file
	FILECONTENTS *f = readFile(argv[1]);
	
	// start the timer
	struct timeval tv1, tv2;
	struct rusage ru_before, ru_after;
	getrusage (RUSAGE_SELF, &ru_before);
	
	int maxDimensions = 0;	//the max number of dimensions in the fronts
	int maxPoints = 0;  //the max number of points in the fronts
	
	// find the max number of Points, and the max number of Dimensions
	for (int i = 0; i < f->nFronts; i++) {
		if (f->fronts[i].nPoints > maxPoints) 
			maxPoints = f->fronts[i].nPoints;
		if (f->fronts[i].n > maxDimensions) 
			maxDimensions = f->fronts[i].n;
  	}

	/* allocate for cuda memory */
	hipMalloc( (void **) &d_temp, maxPoints*maxDimensions*sizeof(float));
	hipMalloc((void **) &d_scanoutput, (512)*sizeof(int));
	hipMalloc((void **) &d_eliminated, (512)*sizeof(int));
	hipMalloc( (void **) &d_front, maxPoints*maxDimensions*sizeof(float));
	hipMalloc((void**) &d_temp2, sizeof(float)*maxPoints*maxDimensions);
	
	// allocate cuda memory
	frontSize = maxPoints*maxDimensions*sizeof(float);
	pointSize = maxDimensions*sizeof(float);
	hipMalloc((void **) &d_frontsArray, frontSize * maxDimensions);
	hipMalloc((void **) &d_ehvStack, sizeof(float) * maxDimensions);
	hipMalloc((void **) &d_hvStack, sizeof(float) * maxDimensions);

	// allocate cpu memory
	indexStack = (int *) malloc(sizeof(int) * maxDimensions);

	// process each front to get the hypervolumes
	for (int i = 0; i < f->nFronts; i++) {
		// read each front
		FRONT front = f->fronts[i];
		n = front.n;

		// copy front to device memory
		float h_front[front.nPoints*n]; 
		for (int j = 0; j < front.nPoints; j++) {
			for (int k = 0; k < n; k++) {
				h_front[j*n+k] = front.points[j].objectives[k];
			}
		}
		hipMemcpy(d_frontsArray, h_front, frontSize, hipMemcpyHostToDevice);		

		// run hv parallel
		hvparallel(front.nPoints);

 		// copy back hvresult
		float hvResult[1];
		hipMemcpy(hvResult, &d_hvStack[0], sizeof(float), hipMemcpyDeviceToHost);

		// print them out
		printf("Calculating Hypervolume for Front:%d...\n", i+1);
		printf("\t\t\t\t\t%1.10f\n", hvResult);
	}
	
	// stop timer
	getrusage (RUSAGE_SELF, &ru_after);
	tv1 = ru_before.ru_utime;
	tv2 = ru_after.ru_utime;
	printf("Average time = %fs\n", (tv2.tv_sec + tv2.tv_usec * 1e-6 - tv1.tv_sec - tv1.tv_usec * 1e-6) / f->nFronts);

	// TODO free the storage

	return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
        exit(-1);
    }                         
}

